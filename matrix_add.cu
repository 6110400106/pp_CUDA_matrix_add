
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 833
void add(int *X, int* Y, int* Z) {

	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			Z[i*N+j] = X[i*N+j] + Y[i*N+j];
		}
	}

}

__global__ void add_kernel(int *X, int *Y, int *Z) {
	
	int i = threadIdx.x;
	int j = threadIdx.y;

	if(i < N && j < N) {
		Z[i*N+j] = X[i*N+j] + Y[i*N+j];
	}
}


int main () {

	//Input matrix
	int X[N*N];
	int Y[N*N];

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			X[i*N+j] = -1;
			Y[i*N+j] = 1;
		}
	}

	//Output matrix
	int Z[N*N];

	int *d_X, *d_Y, *d_Z;
	hipMalloc((void**) &d_X, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Y, (N*N)*sizeof(int));
	hipMalloc((void**) &d_Z, (N*N)*sizeof(int));

	hipMemcpy(d_X, &X, (N*N)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_Y, &Y, (N*N)*sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(32, 32, 1);
	dim3 dimBlock(32, 32, 1);

	//Timed add_kernel function
	hipEventRecord(start);
	add_kernel<<<dimGrid, dimBlock>>>(d_X, d_Y, d_Z);
	hipEventRecord(stop);
	//add(X, Y, Z);

	hipMemcpy(&Z, d_Z, (N*N)*sizeof(int), hipMemcpyDeviceToHost);
	
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	hipFree(d_X);
	hipFree(d_Y);
	hipFree(d_Z);

	int sum = 0;
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			//printf("%d ", Z[i*N+j]);
			sum += Z[i*N+j];
		}
		//printf("\n");
	}
	if(sum == 0)
		printf("All 0s! With N = %d\n", N);
	else {
		printf("Something is wrong!!!\n");
	}
	printf("Time used: %f milliseconds\n", milliseconds);
	
	return -1;

}
